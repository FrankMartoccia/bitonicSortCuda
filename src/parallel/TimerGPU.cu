#include "TimerGPU.cuh"

#include "hip/hip_runtime.h"

// Constructor
TimerGPU::TimerGPU()
{
    hipEventCreate(&m_Start);
    hipEventCreate(&m_Stop);
}

// Starts the timer
void TimerGPU::start() const
{
    hipEventRecord(m_Start);
}

// Stops the timer
void TimerGPU::stop() const
{
    hipEventRecord(m_Stop);
}

// Returns the elapsed time in milliseconds
float TimerGPU::getElapsedMilliseconds() const
{
    hipEventSynchronize(m_Stop);
    float time = 0.0f;
    hipEventElapsedTime(&time, m_Start, m_Stop);
    return time;
}
