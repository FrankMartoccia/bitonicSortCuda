#include "Sort.cuh"
#include "constants.h"
#include <iostream>

#include "TimerGPU.cuh"
#include "bitonicSortGPU.cuh"
#include "hip/hip_runtime.h"

// Error checking function (assuming it is defined somewhere in your project)
void checkCudaError(hipError_t error)
{
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
}

// Constructor: Initialize variables as needed
Sort::Sort(): _d_values(nullptr), _h_values(nullptr), _array_length(0), _sort_order(ORDER_ASC) {
}

// Parameterized Constructor: Initializes member variables
Sort::Sort(uint32_t *d_values, uint32_t *h_values, unsigned int array_length, int sort_order)
    : _d_values(d_values),
      _h_values(h_values),
      _array_length(array_length),
      _sort_order(sort_order){
}

// Method for allocating memory
void Sort::memoryAllocate() {
    // Allocates memory for values on the device
    const hipError_t error = hipMalloc((void **)&_d_values, _array_length * sizeof(*_d_values));
    checkCudaError(error);
}

/*
Memory copy operations needed before sort. If sorting keys only, than "h_values" contains NULL.
*/
void Sort::memoryCopyBeforeSort() const {
    // Copies values
    hipError_t error = hipMemcpy(
        _d_values, _h_values, _array_length * sizeof(*_d_values), hipMemcpyHostToDevice
    );
    checkCudaError(error);
}

void Sort::memoryCopyAfterSort() const {
    // Copies values
    hipError_t error = hipMemcpy(
        _h_values, _d_values, _array_length * sizeof(*_h_values), hipMemcpyDeviceToHost
    );
    checkCudaError(error);
}

// Method for freeing allocated memory
void Sort::memoryFree() const {
    {
        if (_array_length == 0)
        {
            return;
        }

        // Destroy values
        const hipError_t error = hipFree(_d_values);
        checkCudaError(error);
    }
}

/*
Wrapper for bitonic sort method.
The code runs faster if arguments are passed to method. If members are accessed directly, code runs slower.
*/
void Sort::sortValues()
{

    if (_sort_order == ORDER_ASC)
    {
        bitonicSortParallel(_d_values, _array_length, ORDER_ASC);
    }
    else
    {
        bitonicSortParallel(_d_values, _array_length, ORDER_DESC);
    }
}

/*
Wrapper method, which executes all needed memory management and timing. Also calls private sort.
*** Call the constructor first ***
*/
float Sort::sortGPU()
{
    memoryAllocate();

    memoryCopyBeforeSort();

    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);

    TimerGPU timer_gpu;
    timer_gpu.start();
    sortValues();

    error = hipDeviceSynchronize();
    checkCudaError(error);

    timer_gpu.stop();
    const float time = timer_gpu.getElapsedMilliseconds();
    std::cout << "[GPU] - Sorting time: " << time  << " ms" << std::endl;

    memoryCopyAfterSort();

    memoryFree();

    return time;
}
