#include "hip/hip_runtime.h"
#include <iostream>
#include "utils.cuh"
#include "constants.h"
#include "hip/hip_runtime.h"

/*
Executes one step of bitonic merge.
This device function compares and exchanges two elements for each thread.
"offsetGlobal" is the global index offset used for accessing the correct elements.
*/
__device__ void bitonicMergeStep(
    uint32_t *values, unsigned int offsetGlobal, unsigned int arrayLength, unsigned int dataBlockLen, unsigned int stride,
    int sortOrder, int numThreads, bool isFirstStepOfPhase
)
{
    // Each thread will compare and exchange 2 elements in the bitonic merge step
    for (unsigned int tx = threadIdx.x; tx < dataBlockLen >> 1; tx += numThreads)
    {
        unsigned int indexThread = offsetGlobal + tx;
        unsigned int offset = stride;

        // Special handling for the first step of every phase (normalized bitonic sort requires this)
        if (isFirstStepOfPhase)
        {
            // Calculate offset and reverse thread indices within sub-blocks for ascending order
            offset = ((indexThread % stride) * 2) + 1; // "+1 is added to have the offset odd"
            // Recalculate indexThread to mirror its corresponding index within the current stride group.
            // 1. (indexThread / stride) * stride:
            //    - Aligns indexThread to the start of its group (of size 'stride').
            // 2. (indexThread % stride):
            //    - Finds the relative position of indexThread within its group.
            // 3. (stride - 1) - (indexThread % stride):
            //    - Mirrors the thread's position within the group, reversing the order of threads in the current stride.
            indexThread = (indexThread / stride) * stride + ((stride - 1) - (indexThread % stride));
        }

        // Calculate the index used in compareExchange()
        unsigned int index = (indexThread * 2) - (indexThread % stride);

        // Check array bounds to avoid invalid memory access
        if (index + offset >= arrayLength)
        {
            break;
        }

        // Compare and exchange elements based on the sort order
        compareExchange(&values[index], &values[index + offset], sortOrder);
    }
}

/*
Normalized Bitonic Sort Kernel.
This kernel sorts blocks of input data using shared memory for better performance.
*/
__global__ void normalizedBitonicSort(
    uint32_t *valuesGlobal, uint32_t arrayLength, int sortOrder)
{
    // Shared memory to hold the tile being sorted
    extern __shared__ uint32_t bitonicSortTile[];
    unsigned int offset, dataBlockLength;

    // Calculate block-specific data length
    calcDataBlockLength(offset, dataBlockLength, arrayLength, BITONIC_BLOCKS);

    // Copy data from global memory to shared memory
    for (unsigned int tx = threadIdx.x; tx < dataBlockLength; tx += THREADS_BITONIC_SORT)
    {
        bitonicSortTile[tx] = valuesGlobal[offset + tx];
    }
    __syncthreads();

    // Perform the bitonic sorting phases
    for (unsigned int subBlockSize = 1; subBlockSize < dataBlockLength; subBlockSize <<= 1)
    {
        for (unsigned int stride = subBlockSize; stride > 0; stride >>= 1)
        {
            if (stride == subBlockSize)
            {
                // First step of each phase
                bitonicMergeStep(bitonicSortTile, 0, dataBlockLength, dataBlockLength, stride, sortOrder,
                    THREADS_BITONIC_SORT, true);
            }
            else
            {
                // Subsequent steps
                bitonicMergeStep(bitonicSortTile, 0, dataBlockLength, dataBlockLength, stride, sortOrder,
                    THREADS_BITONIC_SORT, false);
            }
            __syncthreads();
        }
    }

    // Copy the sorted data back to global memory
    for (unsigned int tx = threadIdx.x; tx < dataBlockLength; tx += THREADS_BITONIC_SORT)
    {
        valuesGlobal[offset + tx] = bitonicSortTile[tx];
    }
}

/*
Global Bitonic Merge Kernel.
Handles merging of data blocks larger than shared memory.
*/
__global__ void bitonicMergeGlobalKernel(
    uint32_t *dataTable, unsigned int arrayLength, unsigned int step, int sortOrder, bool isFirstStepOfPhase)
{
    unsigned int offset, dataBlockLength;
    calcDataBlockLength(offset, dataBlockLength, arrayLength, MERGE_BLOCKS);

    bitonicMergeStep(
        dataTable, offset / 2, arrayLength, dataBlockLength, 1 << (step - 1), sortOrder,
        THREADS_GLOBAL_MERGE, isFirstStepOfPhase
    );
}

/*
Launches the kernel for bitonic sorting using shared memory.
*/
void runBitonicSortKernel(uint32_t *d_values, unsigned int arrayLength, int sortOrder)
{
    unsigned int elemsPerThreadBlock = arrayLength / BITONIC_BLOCKS;
    unsigned int sharedMemSize = elemsPerThreadBlock * sizeof(*d_values);

    // Define grid and block dimensions for kernel launch
    dim3 dimGrid(BITONIC_BLOCKS, 1, 1);
    dim3 dimBlock(THREADS_BITONIC_SORT, 1, 1);

    // Launch the normalized bitonic sort kernel
    normalizedBitonicSort <<<dimGrid, dimBlock, sharedMemSize>>>(
        d_values, arrayLength, sortOrder
    );
}

/*
Launches the kernel for global bitonic merging.
*/
void runBitonicMergeGlobalKernel(
    uint32_t *d_values, unsigned int arrayLength, unsigned int phase, unsigned int step, int sortOrder)
{
    // unsigned int elemsPerThreadBlock = arrayLength / (THREADS_GLOBAL_MERGE * MERGE_BLOCKS);

    // Define grid and block dimensions for the merge kernel
    dim3 dimGrid(MERGE_BLOCKS, 1, 1);
    dim3 dimBlock(THREADS_GLOBAL_MERGE, 1, 1);

    // Launch the global bitonic merge kernel
    bitonicMergeGlobalKernel <<<dimGrid, dimBlock>>>(
        d_values, arrayLength, step, sortOrder, phase == step
    );
}

/*
Main function to execute parallel bitonic sort on GPU.
*/
void bitonicSortParallel(uint32_t *d_values, unsigned int array_length, int sortOrder)
{
    // Calculate the next power of 2 for the array length
    unsigned int arrayLenPower2 = nextPowerOf2(array_length);
    unsigned int elemsPerBlockBitonicSort = array_length / BITONIC_BLOCKS;

    // Calculate the number of phases for the bitonic sort and merge
    unsigned int phasesBitonicSort = log2(static_cast<double>(min(arrayLenPower2, elemsPerBlockBitonicSort)));
    unsigned int phasesAll = log2(static_cast<double>(arrayLenPower2));

    // Sort sub-blocks of input data using bitonic sort
    runBitonicSortKernel(d_values, array_length, sortOrder);

    // Perform global bitonic merge
    for (unsigned int phase = phasesBitonicSort + 1; phase <= phasesAll; phase++)
    {
        for (unsigned int step = phase; step >= 1; step--)
        {
            runBitonicMergeGlobalKernel(d_values, array_length, phase, step, sortOrder);
        }
    }
}
